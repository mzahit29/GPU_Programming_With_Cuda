#include <iostream>
#include "hip/hip_runtime.h"


using namespace std;

__global__ void kernel(int *a, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < n)
	{
		a[idx] *= 3;
	}
}

int main()
{
	cout << "main() begin" << endl;
	int N = 100000;
	int size = N * sizeof(int);

	int* a_h{};
	a_h = (int *)malloc(size);
	cout << "after malloc. a_h: " << a_h << endl;
	// Fill host memory with some values
	for (int i = 0; i < N; ++i) a_h[i] = i;

	int block_size = 1024; // Count of threads in a block
	// how many blocks required to handle each element of array with one thread
	// we have set thread count per block to be 4.
	int block_count = N / block_size + ((N % block_size) ? 0 : 1);


	hipError_t err;

	int* a_d{};
	hipMalloc(&a_d, size);
	err = hipGetLastError();
	cout << "cudaMalloc: " << hipGetErrorString(err) << endl;

	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	err = hipGetLastError();
	cout << "hipMemcpy host to device: " << hipGetErrorString(err) << endl;


	// Run the program block on the GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	kernel<<< block_count, block_size >>> (a_d, size);
	hipEventRecord(stop);

	// Find the calculation time on GPU
	float calc_time = 0;
	hipEventSynchronize(stop); // Otherwise CPU will not wait for cudaEventElapsedTime to set the calc_time and print 0
	hipEventElapsedTime(&calc_time, start, stop);
	cout << "GPU calculated the result in " << calc_time << " ms" << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	// cudaMemcpy is sync therefore it is inherently a synchronization point
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
	err = hipGetLastError();
	cout << "hipMemcpy device to host: " << hipGetErrorString(err) << endl;
	//cudaDeviceSynchronize();

	// print the result
	//for (int i = 0; i < N; ++i) cout << a_h[i] << endl;

	free(a_h);
	hipFree(a_d);
	cout << "main() end" << endl;
}